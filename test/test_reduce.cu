#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <vector>

#include "block_reduce.hpp"
#include "common.hpp"

template <int blockSize, typename T, typename TAlyxBinaryOp>
__global__ void __launch_bounds__(blockSize) blockReduceKernel(T* a, std::size_t numElement, T* b) {
    unsigned gtid = blockDim.x * blockIdx.x + threadIdx.x;

    T val;
    if (gtid >= numElement) {
        val = TAlyxBinaryOp::init;
    } else {
        val = a[gtid];
    }

    T res = alyx::blockReduce<blockSize>(val, TAlyxBinaryOp{});

    if (threadIdx.x == 0) b[blockIdx.x] = res;
}

template <typename T, typename TAlyxBinaryOp>
class Test {
public:
    void run() {
        CUDA_CHECK(hipSetDevice(0));

        constexpr std::size_t gridSize = 2;
        constexpr std::size_t blockSize = 128;
        std::size_t numElement{200};
        std::vector<T> ah(numElement);

        if constexpr (std::is_same_v<TAlyxBinaryOp, alyx::AlyxBinaryOp::Add<T>>) {
            std::fill(ah.begin(), ah.end(), static_cast<T>(1));
        } else if constexpr (std::is_same_v<TAlyxBinaryOp, alyx::AlyxBinaryOp::Multiply<T>>) {
            std::fill(ah.begin(), ah.end(), static_cast<T>(2));
        } else if constexpr (std::is_same_v<TAlyxBinaryOp, alyx::AlyxBinaryOp::Max<T>> ||
                             std::is_same_v<TAlyxBinaryOp, alyx::AlyxBinaryOp::Min<T>>) {
            std::iota(ah.begin(), ah.end(), static_cast<T>(1));
        }

        T* ad{};
        CUDA_CHECK(hipMalloc(&ad, numElement * sizeof(T)));
        CUDA_CHECK(
            hipMemcpy(ad, ah.data(), numElement * sizeof(T), hipMemcpyKind::hipMemcpyDefault));

        T* bd{};
        CUDA_CHECK(hipMalloc(&bd, gridSize * sizeof(T)));

        blockReduceKernel<blockSize, T, TAlyxBinaryOp><<<gridSize, blockSize>>>(ad, numElement, bd);
        CUDA_CHECK(hipDeviceSynchronize());

        std::vector<T> bh(gridSize);
        CUDA_CHECK(
            hipMemcpy(bh.data(), bd, gridSize * sizeof(T), hipMemcpyKind::hipMemcpyDefault));
        for (std::size_t i = 0; i < bh.size(); ++i) {
            std::cout << bh[i] << " ";
        }
        std::cout << "\n\n";

        CUDA_CHECK(hipFree(ad));
        CUDA_CHECK(hipFree(bd));
    }
};

int main() {
    {
        Test<int, alyx::AlyxBinaryOp::Add<int>> t;
        t.run();
    }

    {
        Test<double, alyx::AlyxBinaryOp::Add<double>> t;
        t.run();
    }

    {
        Test<double, alyx::AlyxBinaryOp::Multiply<double>> t;
        t.run();
    }

    {
        Test<float, alyx::AlyxBinaryOp::Max<float>> t;
        t.run();
    }

    {
        Test<float, alyx::AlyxBinaryOp::Min<float>> t;
        t.run();
    }

    return 0;
}