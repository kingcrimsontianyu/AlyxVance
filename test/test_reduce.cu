#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "block_reduce.hpp"
#include "common.hpp"

enum class OpType {
    Add,
    Multiply,
    Max,
    Min,
};

template <int blockSize, typename T, OpType opType>
__global__ void __launch_bounds__(blockSize) blockReduceKernel(T* a, std::size_t numElement, T* b) {
    unsigned gtid = blockDim.x * blockIdx.x + threadIdx.x;

    T init;
    if constexpr (opType == OpType::Add) {
        init = static_cast<T>(0);
    } else if constexpr (opType == OpType::Multiply) {
        init = static_cast<T>(1);
    } else if constexpr (opType == OpType::Max) {
        init = static_cast<T>(INT_MIN);
    } else if constexpr (opType == OpType::Min) {
        init = static_cast<T>(INT_MAX);
    }

    T val;
    if (gtid >= numElement) {
        val = init;
    } else {
        val = a[gtid];
    }

    T res;
    if constexpr (opType == OpType::Add) {
        res = alyx::blockReduce<blockSize>(val, init, [](T a, T b) { return a + b; });
    } else if constexpr (opType == OpType::Multiply) {
        res = alyx::blockReduce<blockSize>(val, init, [](T a, T b) { return a * b; });
    } else if constexpr (opType == OpType::Max) {
        res = alyx::blockReduce<blockSize>(val, init, [](T a, T b) { return max(a, b); });
    } else if constexpr (opType == OpType::Min) {
        res = alyx::blockReduce<blockSize>(val, init, [](T a, T b) { return min(a, b); });
    }

    if (threadIdx.x == 0) b[blockIdx.x] = res;
}

template <typename T, OpType opType>
class Test {
public:
    void run() {
        CUDA_CHECK(hipSetDevice(0));

        constexpr std::size_t gridSize = 2;
        constexpr std::size_t blockSize = 128;
        std::size_t numElement{200};
        std::vector<T> ah(numElement);

        if constexpr (opType == OpType::Add) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(1);
            }
        } else if constexpr (opType == OpType::Multiply) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(2);
            }
        } else if constexpr (opType == OpType::Max || opType == OpType::Min) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(i);
            }

            std::size_t halfSize = ah.size() >> 1;
            for (std::size_t i = 0; i < halfSize; ++i) {
                if ((i & 1) == 1) {
                    std::swap(ah[i], ah[ah.size() - 1 - i]);
                }
            }
        }

        T* ad{};
        CUDA_CHECK(hipMalloc(&ad, numElement * sizeof(T)));
        CUDA_CHECK(
            hipMemcpy(ad, ah.data(), numElement * sizeof(T), hipMemcpyKind::hipMemcpyDefault));

        T* bd{};
        CUDA_CHECK(hipMalloc(&bd, gridSize * sizeof(T)));

        blockReduceKernel<blockSize, T, opType><<<gridSize, blockSize>>>(ad, numElement, bd);
        CUDA_CHECK(hipDeviceSynchronize());

        std::vector<T> bh(gridSize);
        CUDA_CHECK(
            hipMemcpy(bh.data(), bd, gridSize * sizeof(T), hipMemcpyKind::hipMemcpyDefault));
        for (std::size_t i = 0; i < bh.size(); ++i) {
            std::cout << bh[i] << " ";
        }
        std::cout << "\n";

        CUDA_CHECK(hipFree(ad));
        CUDA_CHECK(hipFree(bd));
    }
};

int main() {
    {
        Test<int, OpType::Add> t;
        t.run();
    }

    // {
    //     Test<double, OpType::Add> t;
    //     t.run();
    // }

    // {
    //     Test<double, OpType::Multiply> t;
    //     t.run();
    // }

    // {
    //     Test<float, OpType::Max> t;
    //     t.run();
    // }

    // {
    //     Test<float, OpType::Min> t;
    //     t.run();
    // }

    return 0;
}