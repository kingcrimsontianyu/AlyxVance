#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#include "block_mergesort.hpp"
#include "common.hpp"

template <int blockSize, typename T, typename Comp>
__global__ void __launch_bounds__(blockSize) blockMergeSortKernel(T* a) {
    unsigned gtid = blockDim.x * blockIdx.x + threadIdx.x;

    a[gtid] = alyx::blockMergeSort<blockSize, T, Comp>(a[gtid], Comp{});
}

template <typename T, typename Comp>
class Test {
public:
    void run() {
        CUDA_CHECK(hipSetDevice(0));

        constexpr std::size_t gridSize = 1;
        constexpr std::size_t blockSize = 128;
        std::size_t numElement{gridSize * blockSize};
        std::vector<T> ah(numElement);

        std::iota(ah.begin(), ah.end(), static_cast<T>(1));

        std::mt19937 rng(2077);
        std::shuffle(ah.begin(), ah.end(), rng);

        T* ad{};
        CUDA_CHECK(hipMalloc(&ad, numElement * sizeof(T)));
        CUDA_CHECK(
            hipMemcpy(ad, ah.data(), numElement * sizeof(T), hipMemcpyKind::hipMemcpyDefault));

        blockMergeSortKernel<blockSize, T, Comp><<<gridSize, blockSize>>>(ad);
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(
            hipMemcpy(ah.data(), ad, numElement * sizeof(T), hipMemcpyKind::hipMemcpyDefault));
        for (std::size_t i = 0; i < ah.size(); ++i) {
            std::cout << ah[i] << " ";
        }
        std::cout << "\n\n";

        CUDA_CHECK(hipFree(ad));
    }
};

int main() {
    {
        Test<int, alyx::Less<int>> t;
        t.run();
    }

    {
        Test<double, alyx::Greater<int>> t;
        t.run();
    }

    return 0;
}