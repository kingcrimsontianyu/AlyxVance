#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "block_seg_scan.hpp"
#include "common.hpp"

enum class OpType {
    Add,
    Multiply,
    Copy,
};

template <int blockSize, typename T, OpType opType>
__global__ void __launch_bounds__(blockSize)
    blockSegScanKernel(T* a, int* flags, std::size_t numElement, T* b) {
    unsigned gtid = blockDim.x * blockIdx.x + threadIdx.x;

    T initVal;
    int initFlag;
    if constexpr (opType == OpType::Add) {
        initVal = alyx::AlyxBinaryOp::SegAdd<T>::init.first;
        initFlag = alyx::AlyxBinaryOp::SegAdd<T>::init.second;
    } else if constexpr (opType == OpType::Multiply) {
        initVal = alyx::AlyxBinaryOp::SegMultiply<T>::init.first;
        initFlag = alyx::AlyxBinaryOp::SegMultiply<T>::init.second;
    } else if constexpr (opType == OpType::Copy) {
        initVal = alyx::AlyxBinaryOp::SegCopy<T>::init.first;
        initFlag = alyx::AlyxBinaryOp::SegCopy<T>::init.second;
    }

    T val;
    int flag;
    if (gtid >= numElement) {
        val = initVal;
        flag = initFlag;
    } else {
        val = a[gtid];
        flag = flags[gtid];
    }

    alyx::SegPair<T> res;
    if constexpr (opType == OpType::Add) {
        res = alyx::blockSegScan<blockSize>(val, flag, alyx::AlyxBinaryOp::SegAdd<T>{});
    } else if constexpr (opType == OpType::Multiply) {
        res = alyx::blockSegScan<blockSize>(val, flag, alyx::AlyxBinaryOp::SegMultiply<T>{});
    } else if constexpr (opType == OpType::Copy) {
        res = alyx::blockSegScan<blockSize>(val, flag, alyx::AlyxBinaryOp::SegCopy<T>{});
    }

    if (gtid < numElement) b[gtid] = res.first;
}

template <typename T, OpType opType>
class Test {
public:
    void run() {
        CUDA_CHECK(hipSetDevice(0));

        constexpr std::size_t gridSize = 2;
        constexpr std::size_t blockSize = 128;
        std::size_t numElement{200};
        std::vector<T> ah(numElement);

        if constexpr (opType == OpType::Add) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(1);
            }
        } else if constexpr (opType == OpType::Multiply) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(2);
            }
        } else if constexpr (opType == OpType::Copy) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(i + 1);
            }
        }

        std::vector<int> flagsH(numElement, 0);
        for (std::size_t i = 0; i < flagsH.size(); ++i) {
            if (i != 0 && (i % 10) == 0) flagsH[i] = 1;
        }

        T* ad{};
        CUDA_CHECK(hipMalloc(&ad, numElement * sizeof(T)));
        CUDA_CHECK(
            hipMemcpy(ad, ah.data(), numElement * sizeof(T), hipMemcpyKind::hipMemcpyDefault));

        int* flagsD{};
        CUDA_CHECK(hipMalloc(&flagsD, numElement * sizeof(int)));
        CUDA_CHECK(hipMemcpy(flagsD, flagsH.data(), numElement * sizeof(int),
                              hipMemcpyKind::hipMemcpyDefault));

        T* bd{};
        CUDA_CHECK(hipMalloc(&bd, numElement * sizeof(T)));

        blockSegScanKernel<blockSize, T, opType>
            <<<gridSize, blockSize>>>(ad, flagsD, numElement, bd);
        CUDA_CHECK(hipDeviceSynchronize());

        std::vector<T> bh(numElement);
        CUDA_CHECK(
            hipMemcpy(bh.data(), bd, numElement * sizeof(T), hipMemcpyKind::hipMemcpyDefault));
        for (std::size_t i = 0; i < bh.size(); ++i) {
            std::cout << bh[i] << " ";
        }
        std::cout << "\n\n";

        CUDA_CHECK(hipFree(ad));
        CUDA_CHECK(hipFree(bd));
    }
};

int main() {
    {
        Test<int, OpType::Add> t;
        t.run();
    }

    {
        Test<int, OpType::Multiply> t;
        t.run();
    }

    {
        Test<double, OpType::Copy> t;
        t.run();
    }

    return 0;
}