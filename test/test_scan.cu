#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "block_scan.hpp"
#include "common.hpp"

enum class OpType {
    Add,
    Multiply,
    Max,
    Min,
};

template <int blockSize, typename T, OpType opType>
__global__ void __launch_bounds__(blockSize) blockScanKernel(T* a, std::size_t numElement, T* b) {
    unsigned gtid = blockDim.x * blockIdx.x + threadIdx.x;

    T init;
    if constexpr (opType == OpType::Add) {
        init = alyx::AlyxBinaryOp::Add<T>::init;
    } else if constexpr (opType == OpType::Multiply) {
        init = alyx::AlyxBinaryOp::Multiply<T>::init;
    } else if constexpr (opType == OpType::Max) {
        init = alyx::AlyxBinaryOp::Max<T>::init;
    } else if constexpr (opType == OpType::Min) {
        init = alyx::AlyxBinaryOp::Min<T>::init;
    }

    T val;
    if (gtid >= numElement) {
        val = init;
    } else {
        val = a[gtid];
    }

    T res;
    if constexpr (opType == OpType::Add) {
        res = alyx::blockScan<blockSize>(val, alyx::AlyxBinaryOp::Add<T>{});
    } else if constexpr (opType == OpType::Multiply) {
        res = alyx::blockScan<blockSize>(val, alyx::AlyxBinaryOp::Multiply<T>{});
    } else if constexpr (opType == OpType::Max) {
        res = alyx::blockScan<blockSize>(val, alyx::AlyxBinaryOp::Max<T>{});
    } else if constexpr (opType == OpType::Min) {
        res = alyx::blockScan<blockSize>(val, alyx::AlyxBinaryOp::Min<T>{});
    }

    if (gtid < numElement) b[gtid] = res;
}

template <typename T, OpType opType>
class Test {
public:
    void run() {
        CUDA_CHECK(hipSetDevice(0));

        constexpr std::size_t gridSize = 2;
        constexpr std::size_t blockSize = 128;
        std::size_t numElement{200};
        std::vector<T> ah(numElement);

        if constexpr (opType == OpType::Add) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(1);
            }
        } else if constexpr (opType == OpType::Multiply) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(2);
            }
        } else if constexpr (opType == OpType::Max || opType == OpType::Min) {
            for (std::size_t i = 0; i < ah.size(); ++i) {
                ah[i] = static_cast<T>(i + 1);
            }
        }

        T* ad{};
        CUDA_CHECK(hipMalloc(&ad, numElement * sizeof(T)));
        CUDA_CHECK(
            hipMemcpy(ad, ah.data(), numElement * sizeof(T), hipMemcpyKind::hipMemcpyDefault));

        T* bd{};
        CUDA_CHECK(hipMalloc(&bd, numElement * sizeof(T)));

        blockScanKernel<blockSize, T, opType><<<gridSize, blockSize>>>(ad, numElement, bd);
        CUDA_CHECK(hipDeviceSynchronize());

        std::vector<T> bh(numElement);
        CUDA_CHECK(
            hipMemcpy(bh.data(), bd, numElement * sizeof(T), hipMemcpyKind::hipMemcpyDefault));
        for (std::size_t i = 0; i < bh.size(); ++i) {
            std::cout << bh[i] << " ";
        }
        std::cout << "\n\n";

        CUDA_CHECK(hipFree(ad));
        CUDA_CHECK(hipFree(bd));
    }
};

int main() {
    {
        Test<int, OpType::Add> t;
        t.run();
    }

    {
        Test<double, OpType::Multiply> t;
        t.run();
    }

    {
        Test<int, OpType::Max> t;
        t.run();
    }

    {
        Test<int, OpType::Min> t;
        t.run();
    }
    return 0;
}